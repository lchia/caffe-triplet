#include "hip/hip_runtime.h"
/*
 * Triplet_loss_layer.cu
 *
 */

#include <vector>

#include "caffe/layers/triplet_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // anchor
      bottom[1]->gpu_data(),  // positive
      diff_ap_.mutable_gpu_data()); // anchor - positive
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // anchor
      bottom[2]->gpu_data(),  // negative
      diff_an_.mutable_gpu_data()); // anchor - negative
  caffe_gpu_sub(
      count,
      bottom[1]->gpu_data(),  // positve
      bottom[2]->gpu_data(),  // negative
      diff_pn_.mutable_gpu_data()); // positive - negative

  caffe_gpu_powx(
      count, 
      diff_an_.mutable_gpu_data(), //anchor_i - negative_i
      Dtype(2),
      diff_sq_an_.mutable_gpu_data()); //(a_i - n_i)^2 
  caffe_gpu_gemv(
      CblasNoTrans, 
      bottom[0]->num(), 
      bottom[0]->channels(), 
      Dtype(1.0),			//alpha 
      diff_sq_an_.gpu_data(),		//A 
      summer_vec_.gpu_data(), 		//x
      Dtype(0.0),			//belta 
      dist_sq_an_.mutable_gpu_data()); // \sum (a_i-n_i)^2   //y

  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  Dtype loss(0.0);
  const Dtype* sampleW = bottom[3]->cpu_data();

  for (int i = 0; i < bottom[0]->num(); ++i) {
      loss += sampleW[i]*std::max(margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i], Dtype(0.0));
  }
  loss = loss / static_cast<Dytpe>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ voidCLLbackward(const int count, const int channels, 
    const Dtype margin, const Dtype alpha, const Dtype* sampleW, 
    const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist-sq-an_, 
    Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(i, count) {
        int n = i / channels; // the num index, to access dist_sq_ap_ and dist_sq_an_
        Dtype mdist(0.0);
        if (mdist > 0.0) {
            bottom_diff[i] = alpha * smapleW[n]*diff[i];
        } else {
            bottom_diff[i] = 0;
        }
    } 
}


template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  const int count = bottom[0]->count();
  const int channels =  bottom[0]->channles();
  
  for (int i = 0; i < 3; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i < 2) ? -1 : 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / static_cast<Dtype>(bottom[i]->num());
      if (i == 0) { 
         //NOLINT_NEXT_LINE(whitespace/operators)
         CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, channels, margin, alpha,
             bottom[3]->gpu_data(),
             diff_pn_.gpu_data(), // the cached eltwise difference between p n
             dist_sq_ap_.gpu_data(), // the cached square difference between a p
             dist_sq_an_.gpu_data(), // the cached square difference between a n
             bottom[i]->mutable_gpu_diff());
         CUDA_POST_KERNEL_CHECK;
      } else if (i == 1) {
         //NOLINT_NEXT_LINE(whitespace/operators)
         CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, channels, margin, alpha,
             bottom[3]->gpu_data(),
             diff_ap_.gpu_data(), // the cached eltwise difference between a p
             dist_sq_ap_.gpu_data(), // the cached square difference between a p
             dist_sq_an_.gpu_data(), // the cached square difference between a n
             bottom[i]->mutable_gpu_diff());
         CUDA_POST_KERNEL_CHECK;
      } else if (i == 2) {
         //NOLINT_NEXT_LINE(whitespace/operators)
         CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, channels, margin, alpha,
             bottom[3]->gpu_data(),
             diff_an_.gpu_data(), // the cached eltwise difference between a n
             dist_sq_ap_.gpu_data(), // the cached square difference between a p
             dist_sq_an_.gpu_data(), // the cached square difference between a n
             bottom[i]->mutable_gpu_diff());
         CUDA_POST_KERNEL_CHECK;
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer); 
}  // namespace caffe
